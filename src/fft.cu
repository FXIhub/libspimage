#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include "spimage.h"


#define cutilSafeCall(err)           __cudaSafeCall      (err, __FILE__, __LINE__)
#define cufftSafeCall(err)           __cufftSafeCall     (err, __FILE__, __LINE__)

inline void __cudaSafeCall( hipError_t err, const char *file, const int line )
{
    if( hipSuccess != err) {
        fprintf(stderr, "cudaSafeCall() Runtime API error in file <%s>, line %i : %s.\n",
                file, line, hipGetErrorString( err) );
        exit(-1);
    }
}

inline void __cufftSafeCall( hipfftResult err, const char *file, const int line )
{
    if( HIPFFT_SUCCESS != err) {
        fprintf(stderr, "cufftSafeCall() CUFFT error in file <%s>, line %i.\n",
                file, line);
        exit(-1);
    }
}

Image * sp_image_cuda_ifft(Image * img){
  hipfftComplex *d_img;
  hipfftHandle plan;
  int size = sp_image_size(img);
  Image * out = sp_image_duplicate(img,SP_COPY_DETECTOR);
  cutilSafeCall(hipMalloc((void**)&d_img, sizeof(hipfftComplex)*size));
  cutilSafeCall(hipMemcpy(d_img, img->image->data, sizeof(hipfftComplex)*size, hipMemcpyHostToDevice));
  if(sp_image_z(img) == 1){
    cufftSafeCall(hipfftPlan2d(&plan, sp_image_x(img),sp_image_y(img), HIPFFT_C2C));
  }else{
    cufftSafeCall(hipfftPlan3d(&plan, sp_image_x(img),sp_image_y(img),sp_image_z(img), HIPFFT_C2C));
  }
  cufftSafeCall(hipfftExecC2C(plan, d_img, d_img, HIPFFT_BACKWARD));
  cutilSafeCall(hipMemcpy(out->image->data, d_img, sizeof(hipfftComplex)*size, hipMemcpyDeviceToHost));
  return out;
}

Image * sp_image_cuda_fft(Image * img){
  hipfftComplex *d_img;
  hipfftHandle plan;
  int size = sp_image_size(img);
  Image * out = sp_image_duplicate(img,SP_COPY_DETECTOR);
  cutilSafeCall(hipMalloc((void**)&d_img, sizeof(hipfftComplex)*size));
  cutilSafeCall(hipMemcpy(d_img, img->image->data, sizeof(hipfftComplex)*size, hipMemcpyHostToDevice));
  if(sp_image_z(img) == 1){
    cufftSafeCall(hipfftPlan2d(&plan, sp_image_x(img),sp_image_y(img), HIPFFT_C2C));
  }else{
    cufftSafeCall(hipfftPlan3d(&plan, sp_image_x(img),sp_image_y(img),sp_image_z(img), HIPFFT_C2C));
  }
  cufftSafeCall(hipfftExecC2C(plan, d_img, d_img, HIPFFT_FORWARD));
  cutilSafeCall(hipMemcpy(out->image->data, d_img, sizeof(hipfftComplex)*size, hipMemcpyDeviceToHost));
  return out;
}
