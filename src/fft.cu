#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include "spimage.h"


Image * sp_image_cuda_ifft(const Image * img){
  hipfftComplex *d_img;
  hipfftHandle plan;
  int size = sp_image_size(img);
  Image * out = sp_image_duplicate(img,SP_COPY_DETECTOR);
  cutilSafeCall(hipMalloc((void**)&d_img, sizeof(hipfftComplex)*size));
  cutilSafeCall(hipMemcpy(d_img, img->image->data, sizeof(hipfftComplex)*size, hipMemcpyHostToDevice));
  if(sp_image_z(img) == 1){
    cufftSafeCall(hipfftPlan2d(&plan, sp_image_y(img),sp_image_x(img), HIPFFT_C2C));
  }else{
    cufftSafeCall(hipfftPlan3d(&plan, sp_image_z(img),sp_image_y(img),sp_image_x(img), HIPFFT_C2C));
  }
  cufftSafeCall(hipfftExecC2C(plan, d_img, d_img, HIPFFT_BACKWARD));
  cutilSafeCall(hipMemcpy(out->image->data, d_img, sizeof(hipfftComplex)*size, hipMemcpyDeviceToHost));
  cufftSafeCall(hipfftDestroy(plan));
  cutilSafeCall(hipFree(d_img));
  return out;
}

Image * sp_image_cuda_fft(const Image * img){
  hipfftComplex *d_img;
  hipfftHandle plan;
  int size = sp_image_size(img);
  Image * out = sp_image_duplicate(img,SP_COPY_DETECTOR);
  cutilSafeCall(hipMalloc((void**)&d_img, sizeof(hipfftComplex)*size));
  cutilSafeCall(hipMemcpy(d_img, img->image->data, sizeof(hipfftComplex)*size, hipMemcpyHostToDevice));
  if(sp_image_z(img) == 1){
    cufftSafeCall(hipfftPlan2d(&plan, sp_image_y(img),sp_image_x(img), HIPFFT_C2C));
  }else{
    cufftSafeCall(hipfftPlan3d(&plan, sp_image_z(img),sp_image_y(img),sp_image_x(img), HIPFFT_C2C));
  }
  cufftSafeCall(hipfftExecC2C(plan, d_img, d_img, HIPFFT_FORWARD));
  cutilSafeCall(hipMemcpy(out->image->data, d_img, sizeof(hipfftComplex)*size, hipMemcpyDeviceToHost));
  cutilSafeCall(hipFree(d_img));
  cufftSafeCall(hipfftDestroy(plan));
  return out;
}
