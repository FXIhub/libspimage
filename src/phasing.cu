#include "hip/hip_runtime.h"
#include <spimage.h>

__global__ void CUDA_module_projection(hipfftComplex* g, const float* amp,const int * pixel_flags, const  int size);
__global__ void CUDA_support_projection_hio(hipfftComplex* g1, const hipfftComplex* g0, const hipfftComplex* gp, const int * pixel_flags,const  int size, const float beta);
__global__ void CUDA_support_projection_er(hipfftComplex* g1, hipfftComplex *gp, const int * pixel_flags, const  int size);
__global__ void CUDA_complex_scale(hipfftComplex * a, int size ,float scale);
__global__ void CUDA_support_projection_raar(hipfftComplex* g1, const hipfftComplex* g0, const hipfftComplex* gp, const int * pixel_flags,const  int size, const float beta);
__global__ void CUDA_apply_constraints(hipfftComplex* g, const int * pixel_flags,const  int size,const SpPhasingConstraints constraints);
__global__ void CUDA_apply_fourier_constraints(hipfftComplex* g, const  int size,const SpPhasingConstraints constraints);
__global__ void CUDA_phased_amplitudes_projection(hipfftComplex* g, const hipfftComplex* phased_amp,const int * pixel_flags, const  int size);
__global__ void CUDA_diff_map_f1(hipfftComplex* f1, const hipfftComplex* g0,const int * pixel_flags,const float gamma1,const  int size);
__global__ void CUDA_diff_map(hipfftComplex* Pi2f1,hipfftComplex* Pi2rho, const hipfftComplex* g0,hipfftComplex* g1,const int * pixel_flags,const float gamma2,const float beta,const  int size);

int sp_proj_module_cuda(Image * a, Image * amp){
  hipfftComplex * d_a;
  int * d_pixel_flags;
  float * d_amp;
  hipMalloc((void **)&d_a,sizeof(hipfftComplex)*sp_image_size(a));
  hipMalloc((void **)&d_pixel_flags,sizeof(int)*sp_image_size(a));
  hipMalloc((void **)&d_amp,sizeof(float)*sp_image_size(a));
  hipMemcpy(d_a,a->image->data,sizeof(hipfftComplex)*sp_image_size(a),hipMemcpyHostToDevice);
  sp_i3matrix * pixel_flags = sp_i3matrix_alloc(sp_image_x(a),sp_image_y(a),sp_image_z(a));
  sp_3matrix * h_amp = sp_3matrix_alloc(sp_image_x(a),sp_image_y(a),sp_image_z(a));
  for(int i =0 ;i<sp_image_size(a);i++){
    h_amp->data[i] = sp_real(amp->image->data[i]);
    pixel_flags->data[i] = 0;
    if(amp->mask->data[i]){
      pixel_flags->data[i] |= SpPixelMeasuredAmplitude;
    }
  }
  hipMemcpy(d_amp,h_amp->data,sizeof(float)*sp_image_size(a),hipMemcpyHostToDevice);
  hipMemcpy(d_pixel_flags,pixel_flags->data,sizeof(int)*sp_image_size(a),hipMemcpyHostToDevice);
  int threads_per_block = 64;
  int number_of_blocks = (sp_image_size(a)+threads_per_block-1)/threads_per_block;
  CUDA_module_projection<<<number_of_blocks, threads_per_block>>>(d_a,d_amp,d_pixel_flags,sp_image_size(a));
  hipMemcpy(a->image->data,d_a,sizeof(hipfftComplex)*sp_image_size(a),hipMemcpyDeviceToHost);
  hipFree(d_amp);
  hipFree(d_pixel_flags);
  hipFree(d_a);
  sp_cuda_check_errors();
  return 0;
}

int phaser_iterate_er_cuda(SpPhaser * ph,int iterations){
  SpPhasingERParameters * params = (SpPhasingERParameters *)ph->algorithm->params;
  for(int i = 0;i<iterations;i++){
    hipfftComplex * swap = ph->d_g0;
    ph->d_g0 = ph->d_g1;
    ph->d_g1 = swap;
    /* executes FFT processes */
    hipfftExecC2C(ph->cufft_plan, ph->d_g0, ph->d_g1, HIPFFT_FORWARD);
    CUDA_apply_fourier_constraints<<<ph->number_of_blocks, ph->threads_per_block>>>(ph->d_g1,ph->image_size,params->constraints);    
    if(ph->phasing_objective == SpRecoverPhases){
      CUDA_module_projection<<<ph->number_of_blocks, ph->threads_per_block>>>(ph->d_g1,ph->d_amplitudes,ph->d_pixel_flags,ph->image_size);
    }else if(ph->phasing_objective == SpRecoverAmplitudes){
      CUDA_phased_amplitudes_projection<<<ph->number_of_blocks, ph->threads_per_block>>>(ph->d_g1,ph->d_phased_amplitudes,ph->d_pixel_flags,ph->image_size);
    }else{
      abort();
    }
    sp_cuda_check_errors();
    sp_cuda_check_errors();
    hipfftExecC2C(ph->cufft_plan, ph->d_g1, ph->d_gp, HIPFFT_BACKWARD);
    /* normalize */
    CUDA_complex_scale<<<ph->number_of_blocks, ph->threads_per_block>>>(ph->d_gp,ph->image_size, 1.0f / (ph->image_size));
    sp_cuda_check_errors();
    CUDA_support_projection_er<<<ph->number_of_blocks, ph->threads_per_block>>>(ph->d_g1, ph->d_gp, ph->d_pixel_flags,ph->image_size);
    sp_cuda_check_errors();
    if(params->constraints != SpNoConstraints){
      CUDA_apply_constraints<<<ph->number_of_blocks, ph->threads_per_block>>>(ph->d_g1,ph->d_pixel_flags,ph->image_size,params->constraints);
    }
    sp_cuda_check_errors();
  }
  ph->iteration += iterations;
  return 0;
}

int phaser_iterate_hio_cuda(SpPhaser * ph,int iterations){
  SpPhasingHIOParameters * params = (SpPhasingHIOParameters *)ph->algorithm->params;
  for(int i = 0;i<iterations;i++){
    real beta = sp_smap_interpolate(params->beta,ph->iteration);
    hipfftComplex * swap = ph->d_g0;
    ph->d_g0 = ph->d_g1;
    ph->d_g1 = swap;
    /* executes FFT processes */
    hipfftExecC2C(ph->cufft_plan, ph->d_g0, ph->d_g1, HIPFFT_FORWARD);
    /* The fourier constraints have to be applied before the amplitude projection otherwise the algorithm never converges,
     probably because there is a deficit of power after the constraints */
    CUDA_apply_fourier_constraints<<<ph->number_of_blocks, ph->threads_per_block>>>(ph->d_g1,ph->image_size,params->constraints);
    sp_cuda_check_errors();
    if(ph->phasing_objective == SpRecoverPhases){
      CUDA_module_projection<<<ph->number_of_blocks, ph->threads_per_block>>>(ph->d_g1,ph->d_amplitudes,ph->d_pixel_flags,ph->image_size);
    }else if(ph->phasing_objective == SpRecoverAmplitudes){
      CUDA_phased_amplitudes_projection<<<ph->number_of_blocks, ph->threads_per_block>>>(ph->d_g1,ph->d_phased_amplitudes,ph->d_pixel_flags,ph->image_size);
    }else{
      abort();
    }
    sp_cuda_check_errors();

    /* The fourier constraints cannot be applied in this location! See comment above */
    hipfftExecC2C(ph->cufft_plan, ph->d_g1, ph->d_gp, HIPFFT_BACKWARD);
    /* normalize */
    CUDA_complex_scale<<<ph->number_of_blocks, ph->threads_per_block>>>(ph->d_gp,ph->image_size, 1.0f / (ph->image_size));
    sp_cuda_check_errors();
    CUDA_support_projection_hio<<<ph->number_of_blocks, ph->threads_per_block>>>(ph->d_g1,ph->d_g0,ph->d_gp,ph->d_pixel_flags,ph->image_size,beta);
    sp_cuda_check_errors();
    if(params->constraints != SpNoConstraints){
      CUDA_apply_constraints<<<ph->number_of_blocks, ph->threads_per_block>>>(ph->d_g1,ph->d_pixel_flags,ph->image_size,params->constraints);
    }
    sp_cuda_check_errors();
    ph->iteration++; 
  }
  return 0;
}

int phaser_iterate_diff_map_cuda(SpPhaser * ph,int iterations){
  SpPhasingDiffMapParameters * params = (SpPhasingDiffMapParameters *)ph->algorithm->params;
  const real gamma1 = params->gamma1;
  const real gamma2 = params->gamma2;
  hipfftComplex * f1;
  hipMalloc((void **)&f1,sizeof(hipfftComplex)*ph->image_size);
  for(int i = 0;i<iterations;i++){
    real beta = sp_smap_interpolate(params->beta,ph->iteration);
    hipfftComplex * swap = ph->d_g0;
    ph->d_g0 = ph->d_g1;
    ph->d_g1 = swap;
    /* executes FFT processes */
    hipfftExecC2C(ph->cufft_plan, ph->d_g0, ph->d_g1, HIPFFT_FORWARD);
    CUDA_apply_fourier_constraints<<<ph->number_of_blocks, ph->threads_per_block>>>(ph->d_g1,ph->image_size,params->constraints);
    CUDA_diff_map_f1<<<ph->number_of_blocks, ph->threads_per_block>>>(f1,ph->d_g0,ph->d_pixel_flags,gamma1,ph->image_size);
    hipfftExecC2C(ph->cufft_plan, f1, f1, HIPFFT_FORWARD);
    CUDA_module_projection<<<ph->number_of_blocks, ph->threads_per_block>>>(f1,ph->d_amplitudes,ph->d_pixel_flags,ph->image_size);
    hipfftExecC2C(ph->cufft_plan, f1, f1, HIPFFT_BACKWARD);
    CUDA_module_projection<<<ph->number_of_blocks, ph->threads_per_block>>>(ph->d_g1,ph->d_amplitudes,ph->d_pixel_flags,ph->image_size);
    hipfftExecC2C(ph->cufft_plan, ph->d_g1, ph->d_gp, HIPFFT_BACKWARD);
    sp_cuda_check_errors();
    /* normalize */
    CUDA_complex_scale<<<ph->number_of_blocks, ph->threads_per_block>>>(ph->d_gp,ph->image_size, 1.0f / (ph->image_size));
    CUDA_complex_scale<<<ph->number_of_blocks, ph->threads_per_block>>>(f1,ph->image_size, 1.0f / (ph->image_size));
    sp_cuda_check_errors();
    CUDA_diff_map<<<ph->number_of_blocks, ph->threads_per_block>>>(f1,ph->d_gp,ph->d_g0,ph->d_g1,ph->d_pixel_flags,gamma2,beta,ph->image_size);
    sp_cuda_check_errors();
    if(params->constraints != SpNoConstraints){
      CUDA_apply_constraints<<<ph->number_of_blocks, ph->threads_per_block>>>(ph->d_g1,ph->d_pixel_flags,ph->image_size,params->constraints);
    }
    sp_cuda_check_errors();
    ph->iteration++; 
  }
  hipFree(f1);
  return 0;
}

int phaser_iterate_raar_cuda(SpPhaser * ph,int iterations){
  SpPhasingRAARParameters * params = (SpPhasingRAARParameters *)ph->algorithm->params;
  for(int i = 0;i<iterations;i++){
    real beta = sp_smap_interpolate(params->beta,ph->iteration);
    hipfftComplex * swap = ph->d_g0;
    ph->d_g0 = ph->d_g1;
    ph->d_g1 = swap;
    /* executes FFT processes */
    hipfftExecC2C(ph->cufft_plan, ph->d_g0, ph->d_g1, HIPFFT_FORWARD);
    sp_cuda_check_errors();
    CUDA_apply_fourier_constraints<<<ph->number_of_blocks, ph->threads_per_block>>>(ph->d_g1,ph->image_size,params->constraints);
    sp_cuda_check_errors();
    if(ph->phasing_objective == SpRecoverPhases){
      CUDA_module_projection<<<ph->number_of_blocks, ph->threads_per_block>>>(ph->d_g1,ph->d_amplitudes,ph->d_pixel_flags,ph->image_size);
    }else if(ph->phasing_objective == SpRecoverAmplitudes){
      CUDA_phased_amplitudes_projection<<<ph->number_of_blocks, ph->threads_per_block>>>(ph->d_g1,ph->d_phased_amplitudes,ph->d_pixel_flags,ph->image_size);
    }else{
      abort();
    }
    sp_cuda_check_errors();
    hipfftExecC2C(ph->cufft_plan, ph->d_g1, ph->d_gp, HIPFFT_BACKWARD);
    /* normalize */
    CUDA_complex_scale<<<ph->number_of_blocks, ph->threads_per_block>>>(ph->d_gp,ph->image_size, 1.0f / (ph->image_size));
    sp_cuda_check_errors();
    CUDA_support_projection_raar<<<ph->number_of_blocks, ph->threads_per_block>>>(ph->d_g1,ph->d_g0,ph->d_gp,ph->d_pixel_flags,ph->image_size,beta);
    sp_cuda_check_errors();
    if(params->constraints != SpNoConstraints){
      CUDA_apply_constraints<<<ph->number_of_blocks, ph->threads_per_block>>>(ph->d_g1,ph->d_pixel_flags,ph->image_size,params->constraints);
    }
    sp_cuda_check_errors();
    ph->iteration++;
  }
  return 0;

}

