#include "hip/hip_runtime.h"
#include <spimage.h>
#include <thrust/reduce.h>
#include <thrust/device_vector.h>

__global__ void CUDA_module_projection(hipfftComplex* g, const float* amp, const float* amp_min, const float* amp_max, const int * pixel_flags,const  int size, const SpPhasingConstraints constraints);
__global__ void CUDA_support_projection_hio(hipfftComplex* g1, const hipfftComplex* g0, const hipfftComplex* gp, const int * pixel_flags,const  int size, const float beta);
__global__ void CUDA_support_projection_er(hipfftComplex* g1, hipfftComplex *gp, const int * pixel_flags, const  int size);
__global__ void CUDA_complex_scale(hipfftComplex * a, int size ,float scale);
__global__ void CUDA_complex_add(hipfftComplex * a, int size ,hipfftComplex add);
__global__ void CUDA_support_projection_raar(hipfftComplex* g1, const hipfftComplex* g0, const hipfftComplex* gp, const int * pixel_flags,const  int size, const float beta);
__global__ void CUDA_apply_constraints(hipfftComplex* g, const int * pixel_flags,const  int size,const SpPhasingConstraints constraints);
__global__ void CUDA_apply_fourier_constraints(hipfftComplex* g, const  int size,const SpPhasingConstraints constraints);
__global__ void CUDA_phased_amplitudes_projection(hipfftComplex* g, const hipfftComplex* phased_amp,const int * pixel_flags, const  int size);
__global__ void CUDA_diff_map_f1(hipfftComplex* f1, const hipfftComplex* g0,const int * pixel_flags,const float gamma1,const  int size);
__global__ void CUDA_diff_map(hipfftComplex* Pi2f1,hipfftComplex* Pi2rho, const hipfftComplex* g0,hipfftComplex* g1,const int * pixel_flags,const float gamma2,const float beta,const  int size);
__global__ void CUDA_random_rephase(hipfftComplex * a, float * uniform_random, int size);
__global__ void CUDA_real_to_complex(hipfftComplex * out, float * in, int size);
__global__ void CUDA_complex_abs2(hipfftComplex * a, int size);
__global__ void CUDA_ereal(hipfftComplex * out, const hipfftComplex * in, const int * pixel_flags, int size);
__global__ void CUDA_efourier(hipfftComplex * out, const hipfftComplex * fmodel, const float* amp, const int * pixel_flags, int size);
__global__ void CUDA_FcFo(hipfftComplex * out, const hipfftComplex * fmodel, const float* amp, const int * pixel_flags, int size);
__global__ void CUDA_pixel_flags_to_complex(hipfftComplex * out, const int * pixel_flags, int size);

struct addCufftComplex{   
  __device__ hipfftComplex operator()(const hipfftComplex lhs, const hipfftComplex rhs) { 
    hipfftComplex temp = lhs;
#ifdef _STRICT_IEEE_754
    temp.x = __fadd_rn(temp.x,rhs.x);
    temp.y = __fadd_rn(temp.y,rhs.y);
#else
    temp.x += rhs.x;
    temp.y += rhs.y;
#endif
    return temp;
  } 
};

static void random_rephase_cuda(SpPhaser * ph, hipfftComplex *  img);

int phaser_iterate_er_cuda(SpPhaser * ph,int iterations){
  SpPhasingERParameters * params = (SpPhasingERParameters *)ph->algorithm->params;
  for(int i = 0;i<iterations;i++){
    hipfftComplex * swap = ph->d_g0;
    ph->d_g0 = ph->d_g1;
    ph->d_g1 = swap;
    /* executes FFT processes */
    hipfftExecC2C(ph->cufft_plan, ph->d_g0, ph->d_g1, HIPFFT_FORWARD);
    CUDA_apply_fourier_constraints<<<ph->number_of_blocks, ph->threads_per_block>>>(ph->d_g1,ph->image_size,params->constraints);    
    if(ph->phasing_objective == SpRecoverPhases){
      CUDA_module_projection<<<ph->number_of_blocks, ph->threads_per_block>>>(ph->d_g1,ph->d_amplitudes,ph->d_amplitudes_min,ph->d_amplitudes_max,ph->d_pixel_flags,ph->image_size,params->constraints);
    }else if(ph->phasing_objective == SpRecoverAmplitudes){
      CUDA_phased_amplitudes_projection<<<ph->number_of_blocks, ph->threads_per_block>>>(ph->d_g1,ph->d_phased_amplitudes,ph->d_pixel_flags,ph->image_size);
    }else{
      abort();
    }
    sp_cuda_check_errors();
    sp_cuda_check_errors();
    hipfftExecC2C(ph->cufft_plan, ph->d_g1, ph->d_gp, HIPFFT_BACKWARD);
    /* normalize */
    CUDA_complex_scale<<<ph->number_of_blocks, ph->threads_per_block>>>(ph->d_gp,ph->image_size, 1.0f / (ph->image_size));
    sp_cuda_check_errors();
    CUDA_support_projection_er<<<ph->number_of_blocks, ph->threads_per_block>>>(ph->d_g1, ph->d_gp, ph->d_pixel_flags,ph->image_size);
    sp_cuda_check_errors();
    if(params->constraints != SpNoConstraints){
      CUDA_apply_constraints<<<ph->number_of_blocks, ph->threads_per_block>>>(ph->d_g1,ph->d_pixel_flags,ph->image_size,params->constraints);
    }
    sp_cuda_check_errors();
  }
  ph->iteration += iterations;
  return 0;
}

int phaser_iterate_hio_cuda(SpPhaser * ph,int iterations){
  SpPhasingHIOParameters * params = (SpPhasingHIOParameters *)ph->algorithm->params;
  for(int i = 0;i<iterations;i++){
    real beta = sp_smap_interpolate(params->beta,ph->iteration);
    hipfftComplex * swap = ph->d_g0;
    ph->d_g0 = ph->d_g1;
    ph->d_g1 = swap;
    /* executes FFT processes */
    hipfftExecC2C(ph->cufft_plan, ph->d_g0, ph->d_g1, HIPFFT_FORWARD);
    /* The fourier constraints have to be applied before the amplitude projection otherwise the algorithm never converges,
     probably because there is a deficit of power after the constraints */
    CUDA_apply_fourier_constraints<<<ph->number_of_blocks, ph->threads_per_block>>>(ph->d_g1,ph->image_size,params->constraints);
    sp_cuda_check_errors();
    if(ph->phasing_objective == SpRecoverPhases){
      CUDA_module_projection<<<ph->number_of_blocks, ph->threads_per_block>>>(ph->d_g1,ph->d_amplitudes,ph->d_amplitudes_min,ph->d_amplitudes_max,ph->d_pixel_flags,ph->image_size,params->constraints);
    }else if(ph->phasing_objective == SpRecoverAmplitudes){
      CUDA_phased_amplitudes_projection<<<ph->number_of_blocks, ph->threads_per_block>>>(ph->d_g1,ph->d_phased_amplitudes,ph->d_pixel_flags,ph->image_size);
    }else{
      abort();
    }
    sp_cuda_check_errors();

    /* The fourier constraints cannot be applied in this location! See comment above */
    hipfftExecC2C(ph->cufft_plan, ph->d_g1, ph->d_gp, HIPFFT_BACKWARD);
    /* normalize */
    CUDA_complex_scale<<<ph->number_of_blocks, ph->threads_per_block>>>(ph->d_gp,ph->image_size, 1.0f / (ph->image_size));
    sp_cuda_check_errors();
    CUDA_support_projection_hio<<<ph->number_of_blocks, ph->threads_per_block>>>(ph->d_g1,ph->d_g0,ph->d_gp,ph->d_pixel_flags,ph->image_size,beta);
    sp_cuda_check_errors();
    if(params->constraints != SpNoConstraints){
      CUDA_apply_constraints<<<ph->number_of_blocks, ph->threads_per_block>>>(ph->d_g1,ph->d_pixel_flags,ph->image_size,params->constraints);
    }
    sp_cuda_check_errors();
    ph->iteration++; 
  }
  return 0;
}

int phaser_iterate_diff_map_cuda(SpPhaser * ph,int iterations){
  SpPhasingDiffMapParameters * params = (SpPhasingDiffMapParameters *)ph->algorithm->params;
  const real gamma1 = params->gamma1;
  const real gamma2 = params->gamma2;
  hipfftComplex * f1;
  hipMalloc((void **)&f1,sizeof(hipfftComplex)*ph->image_size);
  for(int i = 0;i<iterations;i++){
    real beta = sp_smap_interpolate(params->beta,ph->iteration);
    hipfftComplex * swap = ph->d_g0;
    ph->d_g0 = ph->d_g1;
    ph->d_g1 = swap;
    /* executes FFT processes */
    hipfftExecC2C(ph->cufft_plan, ph->d_g0, ph->d_g1, HIPFFT_FORWARD);
    CUDA_apply_fourier_constraints<<<ph->number_of_blocks, ph->threads_per_block>>>(ph->d_g1,ph->image_size,params->constraints);
    CUDA_diff_map_f1<<<ph->number_of_blocks, ph->threads_per_block>>>(f1,ph->d_g0,ph->d_pixel_flags,gamma1,ph->image_size);
    hipfftExecC2C(ph->cufft_plan, f1, f1, HIPFFT_FORWARD);
    CUDA_module_projection<<<ph->number_of_blocks, ph->threads_per_block>>>(f1,ph->d_amplitudes,ph->d_amplitudes_min,ph->d_amplitudes_max,ph->d_pixel_flags,ph->image_size,params->constraints); 
    hipfftExecC2C(ph->cufft_plan, f1, f1, HIPFFT_BACKWARD);
    CUDA_module_projection<<<ph->number_of_blocks, ph->threads_per_block>>>(ph->d_g1,ph->d_amplitudes,ph->d_amplitudes_min,ph->d_amplitudes_max,ph->d_pixel_flags,ph->image_size,params->constraints);
    hipfftExecC2C(ph->cufft_plan, ph->d_g1, ph->d_gp, HIPFFT_BACKWARD);
    sp_cuda_check_errors();
    /* normalize */
    CUDA_complex_scale<<<ph->number_of_blocks, ph->threads_per_block>>>(ph->d_gp,ph->image_size, 1.0f / (ph->image_size));
    CUDA_complex_scale<<<ph->number_of_blocks, ph->threads_per_block>>>(f1,ph->image_size, 1.0f / (ph->image_size));
    sp_cuda_check_errors();
    CUDA_diff_map<<<ph->number_of_blocks, ph->threads_per_block>>>(f1,ph->d_gp,ph->d_g0,ph->d_g1,ph->d_pixel_flags,gamma2,beta,ph->image_size);
    sp_cuda_check_errors();
    if(params->constraints != SpNoConstraints){
      CUDA_apply_constraints<<<ph->number_of_blocks, ph->threads_per_block>>>(ph->d_g1,ph->d_pixel_flags,ph->image_size,params->constraints);
    }
    sp_cuda_check_errors();
    ph->iteration++; 
  }
  hipFree(f1);
  return 0;
}

int phaser_iterate_raar_cuda(SpPhaser * ph,int iterations){
  SpPhasingRAARParameters * params = (SpPhasingRAARParameters *)ph->algorithm->params;
  for(int i = 0;i<iterations;i++){
    real beta = sp_smap_interpolate(params->beta,ph->iteration);
    hipfftComplex * swap = ph->d_g0;
    ph->d_g0 = ph->d_g1;
    ph->d_g1 = swap;
    /* executes FFT processes */
    hipfftExecC2C(ph->cufft_plan, ph->d_g0, ph->d_g1, HIPFFT_FORWARD);
    sp_cuda_check_errors();
    CUDA_apply_fourier_constraints<<<ph->number_of_blocks, ph->threads_per_block>>>(ph->d_g1,ph->image_size,params->constraints);
    sp_cuda_check_errors();
    if(ph->phasing_objective == SpRecoverPhases){
      CUDA_module_projection<<<ph->number_of_blocks, ph->threads_per_block>>>(ph->d_g1,ph->d_amplitudes,ph->d_amplitudes_min,ph->d_amplitudes_max,ph->d_pixel_flags,ph->image_size,params->constraints);
    }else if(ph->phasing_objective == SpRecoverAmplitudes){
      CUDA_phased_amplitudes_projection<<<ph->number_of_blocks, ph->threads_per_block>>>(ph->d_g1,ph->d_phased_amplitudes,ph->d_pixel_flags,ph->image_size);
    }else{
      abort();
    }
    sp_cuda_check_errors();
    hipfftExecC2C(ph->cufft_plan, ph->d_g1, ph->d_gp, HIPFFT_BACKWARD);
    /* normalize */
    CUDA_complex_scale<<<ph->number_of_blocks, ph->threads_per_block>>>(ph->d_gp,ph->image_size, 1.0f / (ph->image_size));
    sp_cuda_check_errors();
    CUDA_support_projection_raar<<<ph->number_of_blocks, ph->threads_per_block>>>(ph->d_g1,ph->d_g0,ph->d_gp,ph->d_pixel_flags,ph->image_size,beta);
    sp_cuda_check_errors();
    if(params->constraints != SpNoConstraints){
      CUDA_apply_constraints<<<ph->number_of_blocks, ph->threads_per_block>>>(ph->d_g1,ph->d_pixel_flags,ph->image_size,params->constraints);
    }
    sp_cuda_check_errors();
    ph->iteration++;
  }
  return 0;

}

int sp_phaser_init_model_cuda(SpPhaser * ph, const Image * user_model, int flags){
  if(!ph){
    return -1;
  }
  if(ph->model){
    sp_image_free(ph->model);
  }
  if(ph->model_change){
    sp_image_free(ph->model_change);
  }
  /* allocate GPU memory */
  cutilSafeCall(hipMalloc((void**)&ph->d_g0, sizeof(hipfftComplex)*ph->image_size));
  cutilSafeCall(hipMalloc((void**)&ph->d_g1, sizeof(hipfftComplex)*ph->image_size));
  cutilSafeCall(hipMalloc((void**)&ph->d_gp, sizeof(hipfftComplex)*ph->image_size));
  cutilSafeCall(hipMalloc((void**)&ph->d_fmodel, sizeof(hipfftComplex)*ph->image_size));

  cutilSafeCall(hipMemset(ph->d_g0, 0, sizeof(hipfftComplex)*ph->image_size));
  cutilSafeCall(hipMemset(ph->d_gp, 0, sizeof(hipfftComplex)*ph->image_size));
  if(ph->nz == 1){
    hipfftPlan2d(&ph->cufft_plan, ph->ny, ph->nx, HIPFFT_C2C);
  }else{
    hipfftPlan3d(&ph->cufft_plan, ph->nz, ph->ny, ph->nx, HIPFFT_C2C);
  }

  ph->model = sp_image_alloc(ph->nx,ph->ny,ph->nz);
  ph->model->phased = 1;
  if(user_model){
    cutilSafeCall(hipMemcpy(ph->d_g1, user_model->image->data, sizeof(hipfftComplex)*ph->image_size, hipMemcpyHostToDevice));
  }else if(flags & SpModelRandomPhases){
    CUDA_real_to_complex<<<ph->number_of_blocks, ph->threads_per_block>>>(ph->d_g1,ph->d_amplitudes, ph->image_size);
    /* randomize phases */
    random_rephase_cuda(ph, ph->d_g1);
    cufftSafeCall(hipfftExecC2C(ph->cufft_plan, ph->d_g1, ph->d_g1, HIPFFT_BACKWARD));
    CUDA_complex_scale<<<ph->number_of_blocks, ph->threads_per_block>>>(ph->d_g1,ph->image_size, 1.0/sp_image_size(ph->model));
  }else if(flags & SpModelZeroPhases){
    cutilSafeCall(hipMemcpy(ph->d_g1,  ph->amplitudes->data, sizeof(hipfftComplex)*ph->image_size, hipMemcpyHostToDevice));
    /* randomize phases */
    cufftSafeCall(hipfftExecC2C(ph->cufft_plan, ph->d_g1, ph->d_g1, HIPFFT_BACKWARD));
    CUDA_complex_scale<<<ph->number_of_blocks, ph->threads_per_block>>>(ph->d_g1,ph->image_size, 1.0/sp_image_size(ph->model));
  }else if(flags & SpModelRandomValues){
    curandSafeCall(hiprandGenerateUniform(ph->gen, (float *)ph->d_g1, ph->image_size*2));
    hipfftComplex add;
    add.x = -0.5;
    add.y = -0.5;
    CUDA_complex_add<<<ph->number_of_blocks, ph->threads_per_block>>>(ph->d_g1,ph->image_size, add);
    /* Note the result does not follow Parseval's theorem as in the CPU code */
  }else{
    return -3;
  }
  if(flags & SpModelMaskedOutZeroed){
    sp_error_fatal("Not implemented in CUDA");
  }
  ph->model->phased = 1;
  ph->model_change = sp_image_alloc(sp_image_x(ph->model),sp_image_y(ph->model),sp_image_z(ph->model));
  cutilSafeCall(hipMemcpy(ph->model->image->data, ph->d_g1, sizeof(hipfftComplex)*ph->image_size, hipMemcpyDeviceToHost));  
  return 0;
}

real sp_phaser_ereal_cuda(SpPhaser * ph){
  /* CUDA_ereal takes the model before projection,  d_gp, and calculate the error pixelwise and stores in d_g0 */
  CUDA_ereal<<<ph->number_of_blocks, ph->threads_per_block>>>(ph->d_g0, ph->d_gp, ph->d_pixel_flags,ph->image_size);
  thrust::device_ptr<hipfftComplex> beginc =  thrust::device_pointer_cast(ph->d_g0);
  thrust::device_ptr<hipfftComplex> endc =  thrust::device_pointer_cast((hipfftComplex *)(ph->d_g0+ph->image_size));
  hipfftComplex sum = {0,0};
  sum = thrust::reduce(beginc,endc,sum,addCufftComplex());
  real ereal = sqrt(sum.x / sum.y);
  return ereal;
}

real sp_phaser_support_fraction_cuda(SpPhaser * ph){
  /* CUDA_ereal takes the model before projection,  d_gp, and calculate the error pixelwise and stores in d_g0 */
  CUDA_pixel_flags_to_complex<<<ph->number_of_blocks, ph->threads_per_block>>>(ph->d_g0, ph->d_pixel_flags,ph->image_size);
  thrust::device_ptr<hipfftComplex> beginc =  thrust::device_pointer_cast(ph->d_g0);
  thrust::device_ptr<hipfftComplex> endc =  thrust::device_pointer_cast((hipfftComplex *)(ph->d_g0+ph->image_size));
  hipfftComplex sum = {0,0};
  sum = thrust::reduce(beginc,endc,sum,addCufftComplex());
  real sup_frac = sum.x / real(ph->image_size);
  return sup_frac;  
}

real sp_phaser_efourier_cuda(SpPhaser * ph, real * FcFo){
  /* Calculate fmodel */
  cufftSafeCall(hipfftExecC2C(ph->cufft_plan, ph->d_g1, ph->d_fmodel, HIPFFT_FORWARD));
  /* CUDA_ereal takes the model before projection,  d_gp, and calculate the error pixelwise and stores in d_g0 */
  CUDA_efourier<<<ph->number_of_blocks, ph->threads_per_block>>>(ph->d_g0, ph->d_fmodel, ph->d_amplitudes, ph->d_pixel_flags,ph->image_size);
  thrust::device_ptr<hipfftComplex> beginc =  thrust::device_pointer_cast(ph->d_g0);
  thrust::device_ptr<hipfftComplex> endc =  thrust::device_pointer_cast((hipfftComplex *)(ph->d_g0+ph->image_size));
  hipfftComplex sum = {0,0};
  sum = thrust::reduce(beginc,endc,sum,addCufftComplex());
  real efourier = sqrt(sum.x / sum.y);
  if(FcFo != NULL){
    /* store the FcFo ratio in this pointer */
    CUDA_FcFo<<<ph->number_of_blocks, ph->threads_per_block>>>(ph->d_g0, ph->d_fmodel, ph->d_amplitudes, ph->d_pixel_flags,ph->image_size);
    sum.x = 0;
    sum.y = 0;
    sum = thrust::reduce(beginc,endc,sum,addCufftComplex());
    *FcFo = sum.x/sum.y;
  }
  return efourier;  
}

static void random_rephase_cuda(SpPhaser * ph, hipfftComplex *  img){
  float * d_uni;
  /* Allocate n floats on device */
  cutilSafeCall(hipMalloc((void **)&d_uni, ph->image_size*sizeof(float)));
  curandSafeCall(hiprandGenerateUniform(ph->gen, d_uni, ph->image_size));
  CUDA_random_rephase<<<ph->number_of_blocks, ph->threads_per_block>>>(img,d_uni,ph->image_size);
}
