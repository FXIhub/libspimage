#include "hip/hip_runtime.h"
#include <spimage.h>

__global__ void CUDA_module_projection(hipfftComplex* g, const float* amp,const int * pixel_flags, const  int size);
__global__ void CUDA_support_projection_hio(hipfftComplex* g1, const hipfftComplex* g0,const int * pixel_flags,const  int size,const float beta);
__global__ void CUDA_support_projection_er(hipfftComplex* g1,const int * pixel_flags,const  int size);
__global__ void CUDA_complex_scale(hipfftComplex * a, int size ,float scale);
__global__ void CUDA_support_projection_raar(hipfftComplex* g1, const hipfftComplex* g0,const int * pixel_flags,const  int size,const float beta);
__global__ void CUDA_apply_constraints(hipfftComplex* g, const int * pixel_flags,const  int size,const SpPhasingConstraints constraints);
__global__ void CUDA_apply_fourier_constraints(hipfftComplex* g, const  int size,const SpPhasingConstraints constraints);
__global__ void CUDA_phased_amplitudes_projection(hipfftComplex* g, const hipfftComplex* phased_amp,const int * pixel_flags, const  int size);
__global__ void CUDA_diff_map_f1(hipfftComplex* f1, const hipfftComplex* g0,const int * pixel_flags,const float gamma1,const  int size);
__global__ void CUDA_diff_map(hipfftComplex* Pi2f1,hipfftComplex* Pi2rho, const hipfftComplex* g0,hipfftComplex* g1,const int * pixel_flags,const float gamma2,const float beta,const  int size);
__global__ void CUDA_ramp_final(hipfftComplex *g, const float ax, const float ay, const int size, const int nx, const int *pixel_flags);

int sp_proj_module_cuda(Image * a, Image * amp){
  hipfftComplex * d_a;
  int * d_pixel_flags;
  float * d_amp;
  hipMalloc((void **)&d_a,sizeof(hipfftComplex)*sp_image_size(a));
  hipMalloc((void **)&d_pixel_flags,sizeof(int)*sp_image_size(a));
  hipMalloc((void **)&d_amp,sizeof(float)*sp_image_size(a));
  hipMemcpy(d_a,a->image->data,sizeof(hipfftComplex)*sp_image_size(a),hipMemcpyHostToDevice);
  sp_i3matrix * pixel_flags = sp_i3matrix_alloc(sp_image_x(a),sp_image_y(a),sp_image_z(a));
  sp_3matrix * h_amp = sp_3matrix_alloc(sp_image_x(a),sp_image_y(a),sp_image_z(a));
  for(int i =0 ;i<sp_image_size(a);i++){
    h_amp->data[i] = sp_real(amp->image->data[i]);
    pixel_flags->data[i] = 0;
    if(amp->mask->data[i]){
      pixel_flags->data[i] |= SpPixelMeasuredAmplitude;
    }
  }
  hipMemcpy(d_amp,h_amp->data,sizeof(float)*sp_image_size(a),hipMemcpyHostToDevice);
  hipMemcpy(d_pixel_flags,pixel_flags->data,sizeof(int)*sp_image_size(a),hipMemcpyHostToDevice);
  int threads_per_block = 64;
  int number_of_blocks = (sp_image_size(a)+threads_per_block-1)/threads_per_block;
  CUDA_module_projection<<<number_of_blocks, threads_per_block>>>(d_a,d_amp,d_pixel_flags,sp_image_size(a));
  hipMemcpy(a->image->data,d_a,sizeof(hipfftComplex)*sp_image_size(a),hipMemcpyDeviceToHost);
  hipFree(d_amp);
  hipFree(d_pixel_flags);
  hipFree(d_a);
  sp_cuda_check_errors();
  return 0;
}

int phaser_iterate_er_cuda(SpPhaser * ph,int iterations){
  SpPhasingERParameters * params = (SpPhasingERParameters *)ph->algorithm->params;
  for(int i = 0;i<iterations;i++){
    hipfftComplex * swap = ph->d_g0;
    ph->d_g0 = ph->d_g1;
    ph->d_g1 = swap;
    /* executes FFT processes */
    hipfftExecC2C(ph->cufft_plan, ph->d_g0, ph->d_g1, HIPFFT_FORWARD);
    CUDA_apply_fourier_constraints<<<ph->number_of_blocks, ph->threads_per_block>>>(ph->d_g1,ph->image_size,params->constraints);    
    if(ph->phasing_objective == SpRecoverPhases){
      CUDA_module_projection<<<ph->number_of_blocks, ph->threads_per_block>>>(ph->d_g1,ph->d_amplitudes,ph->d_pixel_flags,ph->image_size);
    }else if(ph->phasing_objective == SpRecoverAmplitudes){
      CUDA_phased_amplitudes_projection<<<ph->number_of_blocks, ph->threads_per_block>>>(ph->d_g1,ph->d_phased_amplitudes,ph->d_pixel_flags,ph->image_size);
    }else{
      abort();
    }
    sp_cuda_check_errors();
    sp_cuda_check_errors();
    hipfftExecC2C(ph->cufft_plan, ph->d_g1, ph->d_g1, HIPFFT_BACKWARD);
    /* normalize */
    CUDA_complex_scale<<<ph->number_of_blocks, ph->threads_per_block>>>(ph->d_g1,ph->image_size, 1.0f / (ph->image_size));
    sp_cuda_check_errors();
    CUDA_support_projection_er<<<ph->number_of_blocks, ph->threads_per_block>>>(ph->d_g1,ph->d_pixel_flags,ph->image_size);
    sp_cuda_check_errors();
    if(params->constraints != SpNoConstraints){
      CUDA_apply_constraints<<<ph->number_of_blocks, ph->threads_per_block>>>(ph->d_g1,ph->d_pixel_flags,ph->image_size,params->constraints);
    }
    sp_cuda_check_errors();
  }
  ph->iteration += iterations;
  return 0;
}

int phaser_iterate_hio_cuda(SpPhaser * ph,int iterations){
  SpPhasingHIOParameters * params = (SpPhasingHIOParameters *)ph->algorithm->params;
  for(int i = 0;i<iterations;i++){
    real beta = sp_smap_interpolate(params->beta,ph->iteration);
    hipfftComplex * swap = ph->d_g0;
    ph->d_g0 = ph->d_g1;
    ph->d_g1 = swap;
    /* executes FFT processes */
    hipfftExecC2C(ph->cufft_plan, ph->d_g0, ph->d_g1, HIPFFT_FORWARD);
    /* The fourier constraints have to be applied before the amplitude projection otherwise the algorithm never converges,
     probably because there is a deficit of power after the constraints */
    CUDA_apply_fourier_constraints<<<ph->number_of_blocks, ph->threads_per_block>>>(ph->d_g1,ph->image_size,params->constraints);
    sp_cuda_check_errors();
    if(ph->phasing_objective == SpRecoverPhases){
      CUDA_module_projection<<<ph->number_of_blocks, ph->threads_per_block>>>(ph->d_g1,ph->d_amplitudes,ph->d_pixel_flags,ph->image_size);
    }else if(ph->phasing_objective == SpRecoverAmplitudes){
      CUDA_phased_amplitudes_projection<<<ph->number_of_blocks, ph->threads_per_block>>>(ph->d_g1,ph->d_phased_amplitudes,ph->d_pixel_flags,ph->image_size);
    }else{
      abort();
    }
    sp_cuda_check_errors();

    /* The fourier constraints cannot be applied in this location! See comment above */
    hipfftExecC2C(ph->cufft_plan, ph->d_g1, ph->d_g1, HIPFFT_BACKWARD);
    /* normalize */
    CUDA_complex_scale<<<ph->number_of_blocks, ph->threads_per_block>>>(ph->d_g1,ph->image_size, 1.0f / (ph->image_size));
    sp_cuda_check_errors();
    CUDA_support_projection_hio<<<ph->number_of_blocks, ph->threads_per_block>>>(ph->d_g1,ph->d_g0,ph->d_pixel_flags,ph->image_size,beta);
    sp_cuda_check_errors();
    if(params->constraints != SpNoConstraints){
      CUDA_apply_constraints<<<ph->number_of_blocks, ph->threads_per_block>>>(ph->d_g1,ph->d_pixel_flags,ph->image_size,params->constraints);
    }
    sp_cuda_check_errors();
    ph->iteration++; 
  }
  return 0;
}

int phaser_iterate_diff_map_cuda(SpPhaser * ph,int iterations){
  SpPhasingDiffMapParameters * params = (SpPhasingDiffMapParameters *)ph->algorithm->params;
  const real gamma1 = params->gamma1;
  const real gamma2 = params->gamma2;
  hipfftComplex * f1;
  hipMalloc((void **)&f1,sizeof(hipfftComplex)*ph->image_size);
  for(int i = 0;i<iterations;i++){
    real beta = sp_smap_interpolate(params->beta,ph->iteration);
    hipfftComplex * swap = ph->d_g0;
    ph->d_g0 = ph->d_g1;
    ph->d_g1 = swap;
    /* executes FFT processes */
    hipfftExecC2C(ph->cufft_plan, ph->d_g0, ph->d_g1, HIPFFT_FORWARD);
    CUDA_apply_fourier_constraints<<<ph->number_of_blocks, ph->threads_per_block>>>(ph->d_g1,ph->image_size,params->constraints);
    CUDA_diff_map_f1<<<ph->number_of_blocks, ph->threads_per_block>>>(f1,ph->d_g0,ph->d_pixel_flags,gamma1,ph->image_size);
    hipfftExecC2C(ph->cufft_plan, f1, f1, HIPFFT_FORWARD);
    CUDA_module_projection<<<ph->number_of_blocks, ph->threads_per_block>>>(f1,ph->d_amplitudes,ph->d_pixel_flags,ph->image_size);
    hipfftExecC2C(ph->cufft_plan, f1, f1, HIPFFT_BACKWARD);
    CUDA_module_projection<<<ph->number_of_blocks, ph->threads_per_block>>>(ph->d_g1,ph->d_amplitudes,ph->d_pixel_flags,ph->image_size);
    hipfftExecC2C(ph->cufft_plan, ph->d_g1, ph->d_g1, HIPFFT_BACKWARD);
    sp_cuda_check_errors();
    /* normalize */
    CUDA_complex_scale<<<ph->number_of_blocks, ph->threads_per_block>>>(ph->d_g1,ph->image_size, 1.0f / (ph->image_size));
    CUDA_complex_scale<<<ph->number_of_blocks, ph->threads_per_block>>>(f1,ph->image_size, 1.0f / (ph->image_size));
    sp_cuda_check_errors();
    CUDA_diff_map<<<ph->number_of_blocks, ph->threads_per_block>>>(f1,ph->d_g1,ph->d_g0,ph->d_g1,ph->d_pixel_flags,gamma2,beta,ph->image_size);
    sp_cuda_check_errors();
    if(params->constraints != SpNoConstraints){
      CUDA_apply_constraints<<<ph->number_of_blocks, ph->threads_per_block>>>(ph->d_g1,ph->d_pixel_flags,ph->image_size,params->constraints);
    }
    sp_cuda_check_errors();
    ph->iteration++; 
  }
  hipFree(f1);
  return 0;
}

int phaser_iterate_raar_cuda(SpPhaser * ph,int iterations){
  SpPhasingRAARParameters * params = (SpPhasingRAARParameters *)ph->algorithm->params;
  for(int i = 0;i<iterations;i++){
    real beta = sp_smap_interpolate(params->beta,ph->iteration);
    hipfftComplex * swap = ph->d_g0;
    ph->d_g0 = ph->d_g1;
    ph->d_g1 = swap;
    /* executes FFT processes */
    hipfftExecC2C(ph->cufft_plan, ph->d_g0, ph->d_g1, HIPFFT_FORWARD);
    sp_cuda_check_errors();
    CUDA_apply_fourier_constraints<<<ph->number_of_blocks, ph->threads_per_block>>>(ph->d_g1,ph->image_size,params->constraints);
    sp_cuda_check_errors();
    if(ph->phasing_objective == SpRecoverPhases){
      CUDA_module_projection<<<ph->number_of_blocks, ph->threads_per_block>>>(ph->d_g1,ph->d_amplitudes,ph->d_pixel_flags,ph->image_size);
    }else if(ph->phasing_objective == SpRecoverAmplitudes){
      CUDA_phased_amplitudes_projection<<<ph->number_of_blocks, ph->threads_per_block>>>(ph->d_g1,ph->d_phased_amplitudes,ph->d_pixel_flags,ph->image_size);
    }else{
      abort();
    }
    sp_cuda_check_errors();
    hipfftExecC2C(ph->cufft_plan, ph->d_g1, ph->d_g1, HIPFFT_BACKWARD);
    /* normalize */
    CUDA_complex_scale<<<ph->number_of_blocks, ph->threads_per_block>>>(ph->d_g1,ph->image_size, 1.0f / (ph->image_size));
    sp_cuda_check_errors();
    CUDA_support_projection_raar<<<ph->number_of_blocks, ph->threads_per_block>>>(ph->d_g1,ph->d_g0,ph->d_pixel_flags,ph->image_size,beta);
    sp_cuda_check_errors();
    if(params->constraints != SpNoConstraints){
      CUDA_apply_constraints<<<ph->number_of_blocks, ph->threads_per_block>>>(ph->d_g1,ph->d_pixel_flags,ph->image_size,params->constraints);
      if (params->constraints & SpRampObject) {
	apply_ramp_constraint_cuda(ph);
      }
    }
    sp_cuda_check_errors();
    ph->iteration++;
  }
  return 0;

}

int apply_ramp_constraint_cuda(SpPhaser *ph)
{
  float x_tmp, y_tmp;
  float x2,y2,xy,kx,ky;
  Image *tmp = sp_image_alloc(ph->nx,ph->ny,1);
  //int foo = tmp->image->x + tmp->image->y;
  //printf("%d\n",sp_image_y(tmp));
  //printf("%d x %d\n",sp_image_x(tmp),sp_image_y(tmp));
  printf("%d x %d\n",ph->nx,ph->ny);
  hipMemcpy(tmp->image->data,ph->d_g1,sizeof(hipfftComplex)*ph->image_size,hipMemcpyDeviceToHost);
  hipMemcpy(ph->pixel_flags->data,ph->d_pixel_flags,sizeof(int)*ph->image_size,hipMemcpyDeviceToHost);
  int x,y;
  for (x = 0; x < ph->nx/2; x++) {
    x_tmp = (float) x;
    for (y = 0; y < ph->ny/2; y++) {
      if (sp_i3matrix_get(ph->pixel_flags,x,y,0) & SpPixelInsideSupport) {
	y_tmp = (float) y;
	x2 += x_tmp*x_tmp;
	y2 += y_tmp*y_tmp;
	xy += x_tmp*y_tmp;
	kx += x_tmp*sp_carg(sp_image_get(tmp,x,y,0));
	ky += y_tmp*sp_carg(sp_image_get(tmp,x,y,0));
      }
    }
    for (y = ph->ny/2; y < ph->ny; y++) {
      if (sp_i3matrix_get(ph->pixel_flags,x,y,0) & SpPixelInsideSupport) {
	y_tmp = (float) (y-ph->ny);
	x2 += x_tmp*x_tmp;
	y2 += y_tmp*y_tmp;
	xy += x_tmp*y_tmp;
	kx += x_tmp*sp_carg(sp_image_get(tmp,x,y,0));
	ky += y_tmp*sp_carg(sp_image_get(tmp,x,y,0));
      }
    }
  }
  for (x = ph->nx/2; x < ph->nx; x++) {
    x_tmp = (float) (x-ph->nx);
    for (y = 0; y < ph->ny/2; y++) {
      if (sp_i3matrix_get(ph->pixel_flags,x,y,0) & SpPixelInsideSupport) {
	y_tmp = (float) y;
	x2 += x_tmp*x_tmp;
	y2 += y_tmp*y_tmp;
	xy += x_tmp*y_tmp;
	kx += x_tmp*sp_carg(sp_image_get(tmp,x,y,0));
	ky += y_tmp*sp_carg(sp_image_get(tmp,x,y,0));
      }
    }
    for (y = ph->ny/2; y < ph->ny; y++) {
      if (sp_i3matrix_get(ph->pixel_flags,x,y,0) & SpPixelInsideSupport) {
	y_tmp = (float) (y-ph->ny);
	x2 += x_tmp*x_tmp;
	y2 += y_tmp*y_tmp;
	xy += x_tmp*y_tmp;
	kx += x_tmp*sp_carg(sp_image_get(tmp,x,y,0));
	ky += y_tmp*sp_carg(sp_image_get(tmp,x,y,0));
      }
    }
  }
  sp_image_free(tmp);
  /*
  for (int x = 0; x < sp_image_x(new_model); x++) {
    for (int y = 0; y < sp_image_y(new_model); y++) {
      if (sp_i3matrix_get(ph->pixel_flags,x,y,0) & SpPixelInsideSupport) {
	if (x < ph->nx/2) {
	  x_tmp = (real) x;
	} else {
	  x_tmp = (real)( x - ph->nx );
	}
	if (y < ph->ny/2) {
	  y_tmp = (real) y;
	} else {
	  y_tmp = (real)( y - ph->ny );
	}
	x2 += x_tmp*x_tmp;
	y2 += y_tmp*y_tmp;
	xy += x_tmp*y_tmp;
	kx += x_tmp*sp_carg(sp_image_get(new_model,x,y,0));
	ky += y_tmp*sp_carg(sp_image_get(new_model,x,y,0));
      }
    }
  }
  */
  float ax = (kx*y2-ky*xy) / (x2*y2 - xy*xy);
  float ay = (ky*x2-kx*xy) / (x2*y2 - xy*xy);
  printf("\nax = %g\nay = %g\nsize = %d\nnx = %d\n",ax,ay,ph->image_size,ph->nx);

  Image *tmp2 = sp_image_alloc(ph->nx,ph->ny,1);
  hipMemcpy(tmp2->image->data,ph->d_g1,sizeof(hipfftComplex)*ph->image_size,hipMemcpyDeviceToHost);
  sp_image_write(tmp2,"debug_before_kernel.h5",0);

  CUDA_ramp_final<<<ph->number_of_blocks, ph->threads_per_block>>>(ph->d_g1, ax, ay, ph->image_size, ph->nx, ph->d_pixel_flags);

  hipMemcpy(tmp2->image->data,ph->d_g1,sizeof(hipfftComplex)*ph->image_size,hipMemcpyDeviceToHost);
  sp_image_write(tmp2,"debug_after_kernel.h5",0);
  sp_image_free(tmp2);

  //sp_image_free(tmp);
  return 0;
}
